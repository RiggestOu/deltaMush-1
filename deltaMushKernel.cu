#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdio>
__global__ void average_kernel(float * d_in_buffer, float * d_out_buffer, 
                            const int * d_neighbours, const int size)
{
    int s_id = ((blockDim.x * blockIdx.x) +threadIdx.x)*3;
    int d_id =  ((blockDim.x * blockIdx.x) +threadIdx.x)*4;
    
    if(s_id<(size)*3)
    {
        int id;
        float v[3] = {0.0f,0.0f,0.0f};
        for (int i=0; i<4;i++)
        {
            id = d_neighbours[d_id+i]*3;
            v[0] += d_in_buffer[id]; 
            v[1] += d_in_buffer[id+1]; 
            v[2] += d_in_buffer[id+2]; 
        }
        v[0]/= 4.0f;
        v[1]/= 4.0f;
        v[2]/= 4.0f;
        d_out_buffer[s_id] = v[0]; 
        d_out_buffer[s_id+1] = v[1]; 
        d_out_buffer[s_id+2] = v[2]; 
    }
}

__global__ void tangnet_kernel(float * d_smooth, float * d_original, float * d_delta_table)
{
    int s_id = ((blockDim.x * blockIdx.x) +threadIdx.x)*3;
    float v1[3] = {0.0f,0.0f,0.0f};
    float v2[3] = {0.0f,0.0f,0.0f};
    float cross[3] = {0.0f,0.0f,0.0f};
    if(s_id<(size)*3)
    {

        for (int n=0; n<3;n++)
        {

        }
    }
}

void average_launcher(const float * h_in_buffer, float * h_out_buffer, 
                   float * d_in_buffer, float * d_out_buffer, 
                   int * h_neighbours, int* d_neighbours,
                   float * h_delta_table, float * d_delta_table,
                   const int size,int iter)
{
    //copy the memory from cpu to gpu
    int buffer_size = 3*size*sizeof(float);
    
    hipError_t s = hipMemcpy(d_in_buffer, h_in_buffer, buffer_size, hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    
    s = hipMemcpy(d_neighbours, h_neighbours, 4*size*sizeof(int), hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying neigh_table: %s\n", hipGetErrorString(s));
    
    //setup the kernel
    int grain_size =128;
    size_t width_blocks = ((size%grain_size) != 0)?(size/grain_size) +1: (size/grain_size); 
    dim3 block_size(grain_size,1,1);
    dim3 grid_size(width_blocks,1,1);
    
    float * trg= d_in_buffer;
    float * src= d_out_buffer; 
    float * tmp;
    for (int i =0; i<iter; i++)
    {
        tmp = src;
        src = trg;
        trg =tmp; 
        average_kernel<<<grid_size, block_size>>>(src, trg, d_neighbours, size);
    }

    //copy  original data back up
    //if i run the above thread async i might be able to kick this extra memcpy already?
    //to do so I might need another buffer tho
    s = hipMemcpy(d_in_buffer, h_in_buffer, buffer_size, hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    //upload deltas 
    s = hipMemcpy(d_delta_table, h_delta_table, 9*size*sizeof(float), hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    tangnet_kernel<<<grid_size, block_size>>>(d_out_buffer, d_in_buffer, d_delta_table);
    


    //copy data back
    s = hipMemcpy(h_out_buffer, d_out_buffer, 3*size*sizeof(float), hipMemcpyDeviceToHost);
    if (s != hipSuccess) 
            printf("Error copying back: %s\n", hipGetErrorString(s));
}


float * allocate_bufferFloat(int size, int stride)
{
    float * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(float));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}
int * allocate_bufferInt(int size, int stride)
{
    int * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(int));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}

void kernel_tear_down(float * d_in_buffer, float * d_out_buffer, int * d_neigh_table, float * d_delta_table)
{
    if(d_in_buffer);
    {
        hipFree(d_in_buffer);
        d_in_buffer =0;
    }

    if(d_out_buffer)
    {
        hipFree(d_out_buffer);
        d_out_buffer=0;
    }
    
    if(d_neigh_table)
    {
        hipFree(d_neigh_table);
        d_out_buffer = 0;
    }
    if(d_delta_table)
    {
        hipFree(d_delta_table);
        d_out_buffer = 0;
    }
}

