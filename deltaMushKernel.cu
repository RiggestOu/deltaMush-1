#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>


float * allocate_buffer(int size, int stride)
{
    float * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(float));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}

