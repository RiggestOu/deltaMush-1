

#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>
__global__ void average_kernel(float * d_in_buffer, float * d_out_buffer, 
                            const int * d_neighbours, const int size)
{
    int s_id = ((blockDim.x * blockIdx.x) +threadIdx.x)*3;
    int d_id =  ((blockDim.x * blockIdx.x) +threadIdx.x)*4;
    
    if(s_id<(size)*3)
    {
        int id;
        float v[3] = {0.0f,0.0f,0.0f};
        for (int i=0; i<4;i++)
        {
            id = d_neighbours[d_id+i]*3;
            v[0] += d_in_buffer[id]; 
            v[1] += d_in_buffer[id+1]; 
            v[2] += d_in_buffer[id+2]; 
        }
        v[0]/= 4.0f;
        v[1]/= 4.0f;
        v[2]/= 4.0f;
        d_out_buffer[s_id] = v[0]; 
        d_out_buffer[s_id+1] = v[1]; 
        d_out_buffer[s_id+2] = v[2]; 
    }
}

__global__ void tangnet_kernel(float * d_smooth, float * d_original, float * d_delta_table,const int * d_neighbours, const int size)

{
    //id stride 3
    int s_id = ((blockDim.x * blockIdx.x) +threadIdx.x)*3;
    //id stride 4
    int d_id =  ((blockDim.x * blockIdx.x) +threadIdx.x)*4;
    
    //local needed variables 
    float v0[3] = {0.0f,0.0f,0.0f};
    float v1[3] = {0.0f,0.0f,0.0f};
    float v2[3] = {0.0f,0.0f,0.0f};
    float cross[3] = {0.0f,0.0f,0.0f};
    int id;
    
    if(s_id<(size)*3)
    {
        //central vertex
        v0[0] = d_smooth[s_id]; 
        v0[1] = d_smooth[s_id]; 
        v0[2] = d_smooth[s_id]; 

        for (int n=0; n<3;n++)
        {

            id = d_neighbours[d_id+n]*3;
            //first neighbour position
            v1[0] = d_smooth[id]; 
            v1[1] = d_smooth[id+1]; 
            v1[2] = d_smooth[id+2]; 
            
            id = d_neighbours[d_id+n+1]*3;
            //second neighbour position
            v2[0] = d_smooth[id]; 
            v2[1] = d_smooth[id+1]; 
            v2[2] = d_smooth[id+2]; 

            //generate proper vectors
            v1[0] -= v0[0];
            v1[1] -= v0[1];
            v1[2] -= v0[2];
            
            v2[0] -= v0[0];
            v2[1] -= v0[1];
            v2[2] -= v0[2];
            
        }
    }
}

__inline__ __device__ void vec_norm( float * vec)
{

}

void average_launcher(const float * h_in_buffer, float * h_out_buffer, 
                   float * d_in_buffer, float * d_out_buffer, 
                   int * h_neighbours, int* d_neighbours,
                   float * h_delta_table, float * d_delta_table,
                   const int size,int iter)
{
    //copy the memory from cpu to gpu
    int buffer_size = 3*size*sizeof(float);
    
    hipError_t s = hipMemcpy(d_in_buffer, h_in_buffer, buffer_size, hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    
    s = hipMemcpy(d_neighbours, h_neighbours, 4*size*sizeof(int), hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying neigh_table: %s\n", hipGetErrorString(s));
    
    //setup the kernel
    int grain_size =128;
    size_t width_blocks = ((size%grain_size) != 0)?(size/grain_size) +1: (size/grain_size); 
    dim3 block_size(grain_size,1,1);
    dim3 grid_size(width_blocks,1,1);
    
    float * trg= d_in_buffer;
    float * src= d_out_buffer; 
    float * tmp;
    for (int i =0; i<iter; i++)
    {
        tmp = src;
        src = trg;
        trg =tmp; 
        average_kernel<<<grid_size, block_size>>>(src, trg, d_neighbours, size);
    }

    //copy  original data back up
    //if i run the above thread async i might be able to kick this extra memcpy already?
    //to do so I might need another buffer tho
    s = hipMemcpy(d_in_buffer, h_in_buffer, buffer_size, hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    //upload deltas 
    s = hipMemcpy(d_delta_table, h_delta_table, 9*size*sizeof(float), hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    tangnet_kernel<<<grid_size, block_size>>>(d_out_buffer, d_in_buffer, d_delta_table, d_neighbours,size);
    


    //copy data back
    s = hipMemcpy(h_out_buffer, d_out_buffer, 3*size*sizeof(float), hipMemcpyDeviceToHost);
    if (s != hipSuccess) 
            printf("Error copying back: %s\n", hipGetErrorString(s));
}


float * allocate_bufferFloat(int size, int stride)
{
    float * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(float));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}
int * allocate_bufferInt(int size, int stride)
{
    int * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(int));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}

void kernel_tear_down(float * d_in_buffer, float * d_out_buffer, int * d_neigh_table, float * d_delta_table)
{
    if(d_in_buffer);
    {
        hipFree(d_in_buffer);
        d_in_buffer =0;
    }

    if(d_out_buffer)
    {
        hipFree(d_out_buffer);
        d_out_buffer=0;
    }
    
    if(d_neigh_table)
    {
        hipFree(d_neigh_table);
        d_out_buffer = 0;
    }
    if(d_delta_table)
    {
        hipFree(d_delta_table);
        d_out_buffer = 0;
    }
}

